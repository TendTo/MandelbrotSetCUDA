#include "hip/hip_runtime.h"
#include "kernel.cuh"

void error(char const* str)
{
	fprintf(stderr, "%s\n", str);
	exit(1);
}

void cuda_check(hipError_t err, char const* str)
{
	if (err != hipSuccess) {
		fprintf(stderr, "%s: CUDA error %d (%s)\n",
			str, err, hipGetErrorString(err));
	}
}

__global__ void fractal(int nrows, int ncols, int max_iter, double startX, double startY, double zoom, int* result) //373 - 326
{
	const int x_idx = threadIdx.x + blockIdx.x * blockDim.x;
	const int y_idx = threadIdx.y + blockIdx.y * blockDim.y;
	if (x_idx >= ncols || y_idx >= nrows)
		return;

	int l = x_idx + y_idx * ncols;
	double x = ((x_idx - ncols / 2) * zoom + startX) / ncols * 3.5 - 0.75;
	double y = ((y_idx - nrows / 2) * zoom + startY) / nrows * 2.0;
	double re = x, im = y;
	for (int i = 1; i < max_iter; ++i)
	{
		if (re * re + im * im >= 4)
		{
			result[l] = i;
			return;
		}
		double reTemp = re * re - im * im + x;
		im = 2 * re * im + y;
		re = reTemp;
	}
	result[l] = 0;
}

#if __DEBUG
__global__ void fractal_old(int nrows, int ncols, int* result) //398.86642 - 343.53
{
	int x_idx = threadIdx.x + blockIdx.x * blockDim.x;
	int y_idx = threadIdx.y + blockIdx.y * blockDim.y;
	if (x_idx >= nrows || y_idx >= ncols)
		return;

	int l = x_idx * ncols + y_idx;
	double x = (double)x_idx / nrows * 3.5 - 2.5;
	double y = (double)y_idx / ncols * 2.0 - 1.0;
	double re = x, im = y;
	for (int i = 1; i < MAX_ITERATIONS; ++i)
	{
		if (re * re + im * im >= 4)
		{
			result[l] = i;
			return;
		}
		double reTemp = re * re - im * im + x;
		im = 2 * re * im + y;
		re = reTemp;
	}
	result[l] = 0;
}
#endif

int* mendelbrot_kernel(int nrows, int ncols, int max_iter, double startX, double startY, double zoom)
{
	hipError_t err;
	hipEvent_t start, end;
	int* d_result, * h_result;
	size_t res_size = nrows * ncols * sizeof(int);
	float time;

	err = hipMalloc(&d_result, res_size);
	cuda_check(err, "hipMalloc");
	err = hipHostMalloc(&h_result, res_size);
	cuda_check(err, "hipHostMalloc");

	hipEventCreate(&start);
	hipEventCreate(&end);

	dim3 dimBlock(32, 32);
	dim3 dimGrid((ncols + 31) / 32, (nrows + 31) / 32);

	hipEventRecord(start);
	fractal << <dimGrid, dimBlock >> > (nrows, ncols, max_iter, startX, startY, zoom, d_result);
	hipEventRecord(end);


	err = hipMemcpy(h_result, d_result, res_size, hipMemcpyDeviceToHost);
	cuda_check(err, "hipMalloc");

	err = hipEventSynchronize(end);
	cuda_check(err, "sync");

	hipEventElapsedTime(&time, start, end);
	//printf("Tempo passato = %f\n", time);

	err = hipFree(d_result);
	cuda_check(err, "hipFree");

	return h_result;
}

#if __DEBUG
int main()
{
	int* result = kernel_start(12000, 8000);
	//int* result = kernel_start();

	// for (int j = NCOLS - 1; j >= 0; --j)
	// {
	// 	for (int i = 0; i < NROWS; i++)
	// 	{
	// 		printf("%d ", result[i * NCOLS + j]);
	// 	}
	// 	printf("\n");
	// }

	// for (int j = 0; j < NROWS; ++j)
	// {
	// 	for (int i = 0; i < NCOLS; ++i)
	// 	{
	// 		printf("%d ", result[j * NCOLS + i]);
	// 	}
	// 	printf("\n");
	// }
	return 0;
}
#endif